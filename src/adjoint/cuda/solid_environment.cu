#include "hip/hip_runtime.h"


#include <optix_world.h>
#include <cuda/common.hpp>

rtDeclareVariable(float3, color, , );

rtDeclareVariable(RadiancePRD, prd, rtPayload, );

RT_PROGRAM void solid_environment()
{
  prd.result = 0.1f;
}
