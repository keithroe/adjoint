#include "hip/hip_runtime.h"


#include <optix_world.h>
#include <cuda/common.hpp>

using namespace optix;


rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(float,         epsilon, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );

// Should be float4 for max perf.  keeping it simple
rtBuffer<float3, 2>              output_buffer;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );


RT_PROGRAM void pinhole_camera()
{
  float2 d         = make_float2( launch_index ) / 
                     make_float2( launch_dim )   *
                     2.0f - 1.0f;
  float3 origin    = eye;
  float3 direction = normalize(d.x*U + d.y*V + W);
  
  optix::Ray ray   = optix::make_Ray( origin,
                                      direction,
                                      radiance_ray_type,
                                      epsilon,
                                      RT_DEFAULT_MAX);

  RadiancePRD prd;
  prd.result     = 0.0f;
  prd.importance = 1.0f;
  prd.wavelength = 680.0f; // TODO

  rtTrace( top_object, ray, prd );

  output_buffer[ launch_index ] = make_float3( prd.result );
}

